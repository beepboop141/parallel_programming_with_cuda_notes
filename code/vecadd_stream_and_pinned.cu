#include <iostream>
#include <hip/hip_runtime.h>

// Define the kernel for vector addition
__global__ void vec_add(float* A, float* B, float* C, int len) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < len) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int inputLength = 1024 * 1024;  // Length of the vectors
    int streamCount = 4;            // Number of CUDA streams
    int chunkSize = inputLength / streamCount;  // Size of each chunk
    
    float *hostInput1, *hostInput2, *hostOutput;
    float *deviceInput1[streamCount], *deviceInput2[streamCount], *deviceOutput[streamCount];
    hipStream_t streams[streamCount];

    // Allocate pinned memory on the host
    hipHostAlloc((void**)&hostInput1, inputLength * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**)&hostInput2, inputLength * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**)&hostOutput, inputLength * sizeof(float), hipHostMallocDefault);

    // Initialize input vectors with random values
    for (int i = 0; i < inputLength; ++i) {
        hostInput1[i] = static_cast<float>(rand()) / RAND_MAX;
        hostInput2[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate memory on the device and create streams
    for (int i = 0; i < streamCount; ++i) {
        hipMalloc((void**)&deviceInput1[i], chunkSize * sizeof(float));
        hipMalloc((void**)&deviceInput2[i], chunkSize * sizeof(float));
        hipMalloc((void**)&deviceOutput[i], chunkSize * sizeof(float));
        hipStreamCreate(&streams[i]);  // Create CUDA streams
    }

    // Set up execution configuration
    dim3 blockDim(256);
    dim3 gridDim((chunkSize + blockDim.x - 1) / blockDim.x);

    // Process each chunk in a separate stream
    for (int i = 0; i < streamCount; ++i) {
        // Asynchronously copy data from host to device for each stream
        hipMemcpyAsync(deviceInput1[i], hostInput1 + i * chunkSize, chunkSize * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(deviceInput2[i], hostInput2 + i * chunkSize, chunkSize * sizeof(float), hipMemcpyHostToDevice, streams[i]);

        // Launch kernel in each stream
        vec_add<<<gridDim, blockDim, 0, streams[i]>>>(deviceInput1[i], deviceInput2[i], deviceOutput[i], chunkSize);

        // Asynchronously copy results back from device to host for each stream
        hipMemcpyAsync(hostOutput + i * chunkSize, deviceOutput[i], chunkSize * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }

    // Synchronize all streams to make sure all operations are completed
    for (int i = 0; i < streamCount; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    // Validate the result
    bool resultCorrect = true;
    for (int i = 0; i < inputLength; ++i) {
        if (fabs(hostOutput[i] - (hostInput1[i] + hostInput2[i])) > 1e-5) {
            resultCorrect = false;
            break;
        }
    }

    if (resultCorrect) {
        std::cout << "Results are correct!" << std::endl;
    } else {
        std::cout << "Results are incorrect!" << std::endl;
    }

    // Free memory
    for (int i = 0; i < streamCount; ++i) {
        hipFree(deviceInput1[i]);
        hipFree(deviceInput2[i]);
        hipFree(deviceOutput[i]);
        hipStreamDestroy(streams[i]);
    }

    hipHostFree(hostInput1);
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);

    return 0;
}
