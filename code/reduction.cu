#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  // Size of array (multiple of 256 for simplicity)
#define THREADS_PER_BLOCK 256  // Number of threads per block

__global__ void reductionKernel(int *input, int *output, int size) {
    __shared__ int sharedMem[THREADS_PER_BLOCK];

    int tid = threadIdx.x;
    int globalIndex = blockIdx.x * blockDim.x + tid;
    
    // Load data into shared memory (assuming all globalIndex < size)
    if (globalIndex < size)
        sharedMem[tid] = input[globalIndex];
    else
        sharedMem[tid] = 0;

    // Ensure all threads have loaded data into shared memory
    __syncthreads();

    // Perform reduction within each block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();  // Make sure all threads have completed this step
    }

    // Write the result of this block to global memory using atomicAdd
    if (tid == 0) {
        atomicAdd(output, sharedMem[0]);  // Atomic operation to avoid race condition
    }
}

int main() {
    // Size of array
    int size = N * sizeof(int);
    
    // Allocate host memory
    int h_input[N], h_output = 0;
    
    // Initialize input array
    for (int i = 0; i < N; i++) {
        h_input[i] = 1;  // For simplicity, let's set every element to 1
    }

    // Allocate device memory
    int *d_input, *d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, sizeof(int));

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    hipMemcpy(d_output, &h_output, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with (N/THREADS_PER_BLOCK) blocks and THREADS_PER_BLOCK threads
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    reductionKernel<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output, N);

    // Copy result from device to host
    hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    // Output the result
    printf("Sum of array elements: %d\n", h_output);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
